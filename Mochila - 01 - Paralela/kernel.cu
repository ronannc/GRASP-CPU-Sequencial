#include "hip/hip_runtime.h"
/*Bin Packing - two Dimensional
Problema da mochila com duas dimensoes.
Tal problema consiste em escolher o maior numero de itens que comportem em uma mochila (compartimento)

O dataset osado foi o presente na biblioteca ORLIB (OR-Library)

Cada arquivo contem:
Numero de problemas teste
Identificador do Problema
Capacidade do compartimento, numero de itens, numero de itens presentes na melhor solu��o conhecida.
Tamanho dos itens.
*/

#define HAVE_STRUCT_TIMESPEC
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "Objeto.h"
#include "EntradaDeDados.h"
#include "ParallelGRASP.h"
#include "Selection_Sort.h"
#include "SaidaArquivo.h"


typedef struct {
	int max_iter;
	int number_of_itens;
	int bin_capacity;
	item *size_of_itens;
	bool *soluctions;
	int temperatura;
	int decaimento_temperatura;
	int tamanho_RCL;
	int seed;
	int id_thread;
}GRASP_starter;

void* GRASP_thread(void* init)
{
	GRASP_starter arg = *((GRASP_starter* )init);

	//printf("Hello World! It's me, thread with argument %d!\n", (arg).id_thread);

	parallelGRASP((arg).max_iter, (arg).number_of_itens, (arg).bin_capacity, (arg).size_of_itens, (arg).soluctions, (arg).temperatura, (arg).decaimento_temperatura, (arg).tamanho_RCL, (arg).seed, (arg).id_thread);

	return NULL;
}

int main() {
	//semente para gerador de numeros aleatorios
	srand(time(NULL));
	int seed = rand();

	//temperatura inicial e o decaimento da temperatura para SA
	int temperatura = 100;
	int decaimento_temperatura = 1;

	//tamanha usado para tornar a gera��o da solu��o inicial e busca aleatoria, se 1 fica modo guloso
	int tamanho_RCL = 10;

	//numero de itera��es que o GRASP fara
	int max_iter = 200;

	//quantidade de threads e blocos
	const int threads = 5;

	// numero de elementos
	int quantidade_itens = 0;

	// capacidade mochia
	int capacidade_mochila = 0;

	/*recebendo entrada, tamanho de cada item*/
	entrada_dados(quantidade_itens, capacidade_mochila);

	/*Vetor usado para guardar peso e valor e se esta ou nao na mochila*/
	item *itens;
	itens = (item *)malloc(quantidade_itens * sizeof(item));

	if (!itens) {
		printf("Sem memoria disponivel! (itens)\n");
		exit(1);
	}

	entrada_dados_vetor(itens, quantidade_itens);

	//ordenando os dados com rela��o ao valor/peso
	selection_sort(itens, quantidade_itens);

	//for (int i = 0; i < quantidade_itens; i++) {
	//	printf("%d %d\n", itens[i].peso, itens[i].valor);
	//}

	//system("pause");

	printf("====== Bin Packing - Bi Dimensional ======\n");
	printf("\n");
	printf("              IFMG - Formiga            \n");
	printf(" Desenvolvido por: Ronan Nunes Campos   \n");
	printf(" Matricula: 0011919                     \n");
	printf("\n");
	printf(" Dados do problema           \n\n");
	printf(" Numero de Itens: %d                    \n", quantidade_itens);
	printf(" Capacidade Mochila: %d                 \n", capacidade_mochila);
	printf(" Numero de Threads: %d                   \n", threads);
	printf(" Numero de Itera��es: %d                \n", max_iter);
	printf(" Geradas %d soluc�es\n", threads * max_iter);
	printf(" Rodando na CPU                          \n");
	printf("===========================================\n");

	//vetor para guardar o id dos elementos presentes na sulo��o
	bool *soluctions;
	soluctions = (bool *)malloc(quantidade_itens * threads * sizeof(bool));

	if (!soluctions) {
		printf("Sem memoria disponivel! (soluctions)\n");
		exit(1);
	}
	//iniciando a solu��o com todos os itens fora da mochila == 0
	for (int i = 0; i < quantidade_itens * threads; i++) {
		soluctions[i] = 0;
	}

	// Rodando GRASP em paralelo.
	clock_t t0, tf;
	double tempo_gasto;
	int max_valor = 0; int valor = 0; int cont = 0; int aux_id = 0; int cont_id = 0;
	
	pthread_t thread[threads];
	GRASP_starter arg[threads];
	saida_header();

	//for n vezes para executar testes
	for (int k = 0; k < 100; k++) {
		t0 = clock();
		for (int i = 0; i < threads; i++) {
			arg[i] = { max_iter, quantidade_itens, capacidade_mochila, itens, soluctions, temperatura, decaimento_temperatura, tamanho_RCL, seed, i };
			//printf("In main: creating thread %d\n", i);
			int result_code = pthread_create(&thread[i], NULL, GRASP_thread, (void *)&arg[i]);
			assert(0 == result_code);
		}
		for (int i = 0; i < threads; i++) {
			int result_code = pthread_join(thread[i], NULL);
			assert(0 == result_code);
		}
		for (int i = 0; i < quantidade_itens * threads; i++) {

			if (cont < quantidade_itens) {
				if (soluctions[i] == 1) {
					valor += itens[cont].valor;
				}
				cont++;
			}

			if (cont == quantidade_itens) {

				if (valor > max_valor) {
					max_valor = valor;
					aux_id = cont_id;
				}
				cont_id++;

				cont = 0; valor = 0;
			}
		}
		tf = clock();

		tempo_gasto = ((double)(tf - t0)) / CLOCKS_PER_SEC;
		saida_body(max_valor, tempo_gasto, max_iter, threads);
		printf("\n");
		printf("Tempo total gasto: %lf s\n", tempo_gasto);

		printf("===========================================\n\n");
		printf("max valor: %d\n", max_valor);
		for (int i = quantidade_itens * aux_id; i < quantidade_itens * (aux_id + 1); i++) {
			printf("%d ", soluctions[i]);
		}

		printf("\n");
		printf("fim :)\n\n");
	}
	
	free(itens);
	free(soluctions);
	system("pause");
	return 0;
}